#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#define TILE_SIZE 16

__global__ void matrix_multiply(float* matrixA, float* matrixB, float* resultMatrix, int M, int N){
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];
    float sum = 0.0;

    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        if (row < M && t * TILE_SIZE + tx < N) {
            tileA[ty][tx] = matrixA[row * N + t * TILE_SIZE + tx];
        } else {
            tileA[ty][tx] = 0.0;
        }
        if (col < N && t * TILE_SIZE + ty < N) {
            tileB[ty][tx] = matrixB[(t * TILE_SIZE + ty) * N + col];
        } else {
            tileB[ty][tx] = 0.0;
        }
        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += tileA[ty][k] * tileB[k][tx];
        }
        __syncthreads();
    }

    if (row < M && col < N) {
        resultMatrix[row * N + col] = sum;
    }
}

int main(){
    int M = 200;
    int N = 100;
    float* host_matrixA, * host_matrixB, * host_resultMatrix;
    float* device_matrixA, * device_matrixB, * device_resultMatrix;
    int size_matrixA = M * N * sizeof(float);
    int size_matrixB = N * N * sizeof(float);
    int size_resultMatrix = M * N * sizeof(float);

    host_matrixA = (float*)malloc(size_matrixA);
    host_matrixB = (float*)malloc(size_matrixB);
    host_resultMatrix = (float*)malloc(size_resultMatrix);

    srand(time(NULL));
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            host_matrixA[i * N + j] =rand() % 10 + 1;
        }
    }
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            host_matrixB[i * N + j] =rand() % 10 + 1;
        }
    }

    hipMalloc(&device_matrixA, size_matrixA);
    hipMalloc(&device_matrixB, size_matrixB);
    hipMalloc(&device_resultMatrix, size_resultMatrix);

    hipMemcpy(device_matrixA, host_matrixA, size_matrixA, hipMemcpyHostToDevice);
    hipMemcpy(device_matrixB, host_matrixB, size_matrixB, hipMemcpyHostToDevice);

    dim3 dimGrid((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE, 1);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matrix_multiply<<<dimGrid, dimBlock>>>(device_matrixA, device_matrixB, device_resultMatrix, M, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(host_resultMatrix, device_resultMatrix, size_resultMatrix, hipMemcpyDeviceToHost);

    printf("Execution time: %.2f ms\n", milliseconds);

    free(host_matrixA);
    free(host_matrixB);
    free(host_resultMatrix);
    hipFree(device_matrixA);
    hipFree(device_matrixB);
    hipFree(device_resultMatrix);

    return 0;
}
