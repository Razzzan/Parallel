#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel function for matrix multiplication
__global__ void matrix_multiply(float *matrixA, float *matrixB, float *resultMatrix, int rows, int columns){
    int row_index = blockIdx.x * blockDim.x + threadIdx.x;
    int col_index = blockIdx.y * blockDim.y + threadIdx.y;

    if (row_index < rows && col_index < columns) {
        float sum = 0.0;
        for (int k = 0; k < columns; k++) {
            sum += matrixA[row_index * columns + k] * matrixB[k * columns + col_index];
        }
        resultMatrix[row_index * columns + col_index] = sum;
    }
}

int main(){
    int num_rows = 500;
    int num_columns = 250;
    float *host_matrixA, *host_matrixB, *host_resultMatrix;
    float *device_matrixA, *device_matrixB, *device_resultMatrix;

    int size_matrixA = num_rows * num_columns * sizeof(float);
    int size_matrixB = num_columns * num_columns * sizeof(float);
    int size_resultMatrix = num_rows * num_columns * sizeof(float);

    // Allocate memory on host
    host_matrixA = (float*) malloc(size_matrixA);
    host_matrixB = (float*) malloc(size_matrixB);
    host_resultMatrix = (float*) malloc(size_resultMatrix);

    // Initialize matrices
    for (int i = 0; i < num_rows; i++) {
        for (int j = 0; j < num_columns; j++) {
            host_matrixA[i * num_columns + j] = i + j;
        }
    }
    for (int i = 0; i < num_columns; i++) {
        for (int j = 0; j < num_columns; j++) {
            host_matrixB[i * num_columns + j] = i - j;
        }
    }

    // Allocate memory on device
    hipMalloc(&device_matrixA, size_matrixA);
    hipMalloc(&device_matrixB, size_matrixB);
    hipMalloc(&device_resultMatrix, size_resultMatrix);

    // Copy data from host to device
    hipMemcpy(device_matrixA, host_matrixA, size_matrixA, hipMemcpyHostToDevice);
    hipMemcpy(device_matrixB, host_matrixB, size_matrixB, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 grid_dim((num_rows + 15) / 16, (num_columns + 15) / 16, 1);
    dim3 block_dim(16, 16, 1);

    // Record start time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch kernel for matrix multiplication
    matrix_multiply<<<grid_dim, block_dim>>>(device_matrixA, device_matrixB, device_resultMatrix, num_rows, num_columns);

    // Record stop time and synchronize device
    hipEventRecord(stop);
    hipDeviceSynchronize();

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy result back to host
    hipMemcpy(host_resultMatrix, device_resultMatrix, size_resultMatrix, hipMemcpyDeviceToHost);

    // Print execution time in milliseconds
    printf("Execution time: %.2f ms\n", milliseconds);

    // Free memory
    free(host_matrixA);
    free(host_matrixB);
    free(host_resultMatrix);
    hipFree(device_matrixA);
    hipFree(device_matrixB);
    hipFree(device_resultMatrix);

    return 0;
}
